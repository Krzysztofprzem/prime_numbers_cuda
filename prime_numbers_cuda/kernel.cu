#include "hip/hip_runtime.h"
#include <iostream>
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include <time.h>
#include <cstdlib>

using namespace std;
hipError_t addWithCuda(long long unsigned int liczba, bool *pierwsza);


__global__ void PierwszaCzyZlozona23(long long unsigned int *liczba, bool *pierwsza)
{
	long long unsigned int index = threadIdx.x;
	if (*liczba % (index + 2) == 0)
		*pierwsza = false;
}

__global__ void PierwszaCzyZlozona(long long unsigned int *liczba, bool *pierwsza, long long unsigned int *przesuniecie)
{
	long long unsigned int i = (threadIdx.x + blockDim.x*blockIdx.x + *przesuniecie) * 6;
	if (*liczba % ((i + 5)) == 0) { *pierwsza = false; }
	if (*liczba % ((i + 5) + 2) == 0) { *pierwsza = false; }
}

int main()
{
	// PIERWSZE
	// 2^31-1 = 2147483647
	// 2^61-1 = 2305843009213693951
	// ZLOZONE
	// (2^31-1)^2		= 4611686014132420609
	// (2^31-1)(2^13-1) = 17590038552577


	unsigned long long int liczba = 0;
	bool pierwsza = true;
	time_t startCPU;
	time_t stopCPU;
	time_t startGPU;
	time_t stopGPU;

	cout << "Podaj liczbe" << endl;
	cin >> liczba;

	cout << "SPRAWDZANIE DLA CPU" << endl;
	startCPU = clock();

	if (liczba % 2 == 0) pierwsza = false;
	else if (liczba % 3 == 0) pierwsza = false;

	if (pierwsza)
		for (unsigned long long int i = 5; i <= sqrt(liczba); i = i + 6) {
			if (liczba % i == 0) { pierwsza = false; break; }
			if (liczba % (i + 2) == 0) { pierwsza = false; break; }
		}

	if (pierwsza) {
		cout << "Liczba pierwsza" << endl;
	}
	else {
		cout << "Liczba zlozona" << endl;
	}

	stopCPU = clock();
	double czasCPU = (double)(stopCPU - startCPU) / CLOCKS_PER_SEC;
	cout << "Czas sprawdzania na CPU wynosi: " << czasCPU << endl;

	pierwsza = true;
	cout << "SPRAWDZANIE DLA GPU" << endl;
	startGPU = clock();
	hipError_t cudaStatus = addWithCuda(liczba, &pierwsza);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		char l;
		cin >> l;
		return 1;
	}

	if (pierwsza) {
		cout << "Liczba pierwsza" << endl;
	}
	else {
		cout << "Liczba zlozona" << endl;
	}

	stopGPU = clock();
	double czasGPU = (double)(stopGPU - startGPU) / CLOCKS_PER_SEC;
	cout << "Czas sprawdzania na GPU wynosi: " << czasGPU << endl;

	double przyspieszenie = (double)(czasCPU / czasGPU);
	cout << "Przyspieszenie na GPU wzgl�dem CPU: " << przyspieszenie << endl;
	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	system("pause");

	return 0;
}

hipError_t addWithCuda(long long unsigned int liczba, bool *pierwsza)
{
	long long unsigned int *dev_liczba = 0;
	bool				   *dev_pierwsza = 0;
	long long unsigned int	przesuniecie = 0;
	long long unsigned int *dev_przesuniecie = 0;

	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_liczba, sizeof(long long unsigned int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_pierwsza, sizeof(bool));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_liczba, &liczba, sizeof(long long unsigned int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_pierwsza, pierwsza, sizeof(bool), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// SPRAWDZENIE PATOLOGICZNYCH PRZYPADKOW
	PierwszaCzyZlozona23 << <1, 2 >> > (dev_liczba, dev_pierwsza);

	if (*pierwsza) {
		// PRZYGOTOWANIE DO PODZIALU NA SIATKI BLOKOW I BLOKI WATKOW
		int ilosc_watkow_w_bloku = 1024;
		int ilosc_blokow_w_siatce = 65535;
		long long unsigned int ilosc_iteracji = (sqrt(liczba) + 1) / 6;
		//cout << "ilosc iteracji  " << ilosc_iteracji << endl;
		long long unsigned int ilosc_pelnych_blokow = ilosc_iteracji / ilosc_watkow_w_bloku;
		//cout << "ilosc_pelnych_blokow  " << ilosc_pelnych_blokow << endl;
		long long unsigned int ilosc_watkow_w_niepelnym_bloku = ilosc_iteracji % ilosc_watkow_w_bloku;
		//cout << "ilosc_watkow_w_niepelnym_bloku  " << ilosc_watkow_w_niepelnym_bloku << endl;
		long long unsigned int ilosc_blokow = (ilosc_watkow_w_niepelnym_bloku == 0) ? ilosc_pelnych_blokow : ilosc_pelnych_blokow + 1;
		//cout << "ilosc_blokow  " << ilosc_blokow << endl;

		long long unsigned int ilosc_pelnych_siatek = ilosc_blokow / ilosc_blokow_w_siatce;
		//cout << "ilosc_pelnych_siatek  " << ilosc_pelnych_siatek << endl;
		long long unsigned int ilosc_blokow_w_niepelnej_siatce = ilosc_blokow % ilosc_blokow_w_siatce;
		//cout << "ilosc_blokow_w_niepelnej_siatce  " << ilosc_blokow_w_niepelnej_siatce << endl;
		long long unsigned int ilosc_siatek = (ilosc_blokow_w_niepelnej_siatce == 0) ? ilosc_pelnych_siatek : ilosc_pelnych_siatek + 1;
		//cout << "ilosc_siatek  " << ilosc_siatek << endl;

		cudaStatus = hipMalloc((void**)&dev_przesuniecie, sizeof(long long unsigned int));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}

		cudaStatus = hipMemcpy(dev_przesuniecie, &przesuniecie, sizeof(long long unsigned int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

		for (long long unsigned int i = 0; i < ilosc_siatek; i++) {
			przesuniecie = i * ilosc_blokow_w_siatce*ilosc_watkow_w_bloku;
			cudaStatus = hipMemcpy(dev_przesuniecie, &przesuniecie, sizeof(long long unsigned int), hipMemcpyHostToDevice);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMemcpy failed!");
				goto Error;
			}

			if (i == ilosc_siatek - 1) {
				// PRZYPADEK W KTORYM BADAMY NIEPELNA SIATKE Z PELNYMI BLOKAMI
				if (ilosc_blokow_w_niepelnej_siatce > 1) {
					PierwszaCzyZlozona << <ilosc_blokow_w_niepelnej_siatce - 1, ilosc_watkow_w_bloku >> > (dev_liczba, dev_pierwsza, dev_przesuniecie);
					if (!*pierwsza)
						break;

					przesuniecie += (ilosc_blokow_w_niepelnej_siatce - 1)*ilosc_watkow_w_bloku;
					cudaStatus = hipMemcpy(dev_przesuniecie, &przesuniecie, sizeof(long long unsigned int), hipMemcpyHostToDevice);
					if (cudaStatus != hipSuccess) {
						fprintf(stderr, "hipMemcpy failed!");
						goto Error;
					}
				}
				// PRZYPADEK W KTORYM BADAMY NIEPELNY BLOK
				PierwszaCzyZlozona << <1, ilosc_watkow_w_niepelnym_bloku >> > (dev_liczba, dev_pierwsza, dev_przesuniecie);
			}
			else
				// PRZYPADEK W KTORYM BADAMY PELNA SIATKE Z PELNYMI BLOKAMI
				PierwszaCzyZlozona << <ilosc_blokow_w_siatce, ilosc_watkow_w_bloku >> > (dev_liczba, dev_pierwsza, dev_przesuniecie);

			if (!*pierwsza)
				break;
		}
	}

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	cudaStatus = hipMemcpy(pierwsza, dev_pierwsza, sizeof(bool), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}


Error:

	hipFree(dev_liczba);
	hipFree(dev_pierwsza);
	hipFree(dev_przesuniecie);

	return cudaStatus;
}

